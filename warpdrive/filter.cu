/*
filter.cu contains various convolution functions for the detector class to call.

Andrew Barentine - Spring 2016
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dog_row_variance_convolution(float *var, float *rconvdata, float *filter,// const int rowsize,
int halfFilt, const int colsize)
/*
This function takes input data and performs a row convolution. The convolution is stored in a separate
output array, and is performed on the inverse of the incoming data (variance).

Each row is loaded into shared memory before the convolution is performed. Currently, the maximum size array that can
be convolved by this function is 1024x1024, because each pixel is assigned its own thread.
*/
{
    int k, halfFiltm1 = halfFilt-1;
    int rid = blockIdx.x;// + halfFilt;
    int j = threadIdx.x;// + halfFilt;
    float tempsum = 0;

    volatile __shared__ float rdata_sh[1075]; //FIXME: should be changed to colsize (PADDED SIZE, or larger)
    __shared__ float filter_sh[12];

    // pad row of shared memory
    if (j < (halfFilt)){
        rdata_sh[j] = 0;
        rdata_sh[colsize + j + halfFilt] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    if ((j == (halfFilt-1)) && (rid == 1)){
        printf("For Gondor!!");
    }
    // load row of data into shared mem and weight on variance
    rdata_sh[j + halfFilt] = 1/var[rid*colsize + j];

    // Load filter into shared memory
    if (j < (2*halfFilt)) filter_sh[j] = filter[j];

    // make sure we've loaded everything we need to
    __syncthreads();

    //if ((j==colsize-1)&&(rid==5)) printf("colsize + halfFilt %d", (colsize + j));

    // Perform convolution
    for (k = -halfFilt; k <= halfFiltm1; k++){
        tempsum += rdata_sh[(j + halfFilt) - k]*filter_sh[k + halfFilt];
    }
    // push to output array
    rconvdata[rid*colsize + j] = tempsum;

}


__global__ void convRowGPU_comb(float *data, float *var, float *rowConvBig, float *rowConvSmall,
float *filterBig, float *filterSmall, int halfFiltBig, int halfFiltSmall, int colsize)
/*
THIS FUNCTION HAS BEEN DEPRECIATED, as for some reason the maximum array size it can convolve is less than 1024x1024
This function takes input data and performs TWO row convolutions. The convolutions are stored in separate
output arrays.

Each row is loaded into shared memory before the convolution is performed.
*/
{
    int k;
    int rid = blockIdx.x;// + threadIdx.y;
    int j = threadIdx.x;
    float tempsum = 0;

    //printf("threadIdx.y %d", threadIdx.y);

    volatile __shared__ float rdata_sh[1075]; //must be padded-image size + 1, or larger)
    //volatile __shared__ float rdata2_sh[275];
    __shared__ float filterBig_sh[12]; //must be at least the filter size
    __shared__ float filterSmall_sh[12]; //must be at least the filter size

    if (j < (halfFiltBig)){
        rdata_sh[j] = 0;
        rdata_sh[colsize + j + halfFiltBig] = 0;
        //printf("halfFiltBig, halfFiltSmall %d", halfFiltSmall);
    }
    rdata_sh[j + halfFiltBig] = data[rid*colsize + j]/var[rid*colsize + j];
    // Note that in the future, these next two lines could be combined by sending
    // in the filters concatenated into a single array
    if (j < (2*halfFiltBig)){
        filterBig_sh[j] = filterBig[j];
    }
    if (j < (2*halfFiltSmall)){
        filterSmall_sh[j] = filterSmall[j];
    }

    __syncthreads(); //make sure each thread is finished writing to shared memory

    //printf("threadIdx.y %d", threadIdx.y);
    for (k = -halfFiltBig; k < halfFiltBig; k++){
        tempsum += rdata_sh[(j + halfFiltBig) - k]*filterBig_sh[k + halfFiltBig];
    }
    rowConvBig[rid*colsize + j] = tempsum;
    //rowConvBig[rid*colsize + j] = rdata_sh[j + halfFiltBig][threadIdx.y];

    tempsum = 0;

    for (k = -halfFiltSmall; k <= halfFiltSmall; k++){
        tempsum += rdata_sh[(j + halfFiltBig) - k]*filterSmall_sh[k + halfFiltSmall];
    }
    rowConvSmall[rid*colsize + j] = tempsum;
    //rowConvSmall[rid*colsize + j] = rdata_sh[j + halfFiltBig][threadIdx.y];

}

__global__ void dog_row_convolution(float *data, float *var, float *row_convolved_data, float *filter,// const int rowsize,
int half_filter_size, float *background)
/*
    Perform the first part of a separable convolution. FIXME - finish this description

    Parameters
    ----------
    data: input data, camera-corrected and converted to units of e-
    var: (per-pixel) variance due to readout noise [e-^2]
    rconvdata: memory allocation to store result


    CUDA indexing
    -------------
    block
        x: n_columns
            size[1] of the variance map
    grid
        x: n_rows
            size[0] of the variance map

    Notes
    -----
    Note that the PYME.remFitBuf.fitTask.calcSigma returns variance in [ADU^2] while here we return in e-^2
*/
/*
This function takes input data, subtracts the pixel-dependent background estimate, converts the data from units of ADU
to photoelectrons and performs a row convolution. The convolution is stored in a separate output array.
Each row is loaded into shared memory before the convolution is performed. Currently, the maximum size array that can
be convolved by this function is 1024x1024, because each pixel is assigned its own thread.
*/
{
    int k;
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    float temp_sum = 0;

    volatile __shared__ float rdata_sh[1075]; //should be changed to blockDim.x (PADDED SIZE, or larger)
    __shared__ float filter_sh[12];

    // Pad the shared memory array
    if (threadIdx.x < (half_filter_size)){
        rdata_sh[threadIdx.x] = 0;
        rdata_sh[blockDim.x + threadIdx.x + half_filter_size] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    // load row of data into shared mem and subtract background
    rdata_sh[threadIdx.x + half_filter_size] = (data[ind] - background[ind])/var[ind];
    if (threadIdx.x < (2 * half_filter_size)) filter_sh[threadIdx.x] = filter[threadIdx.x];

    // make sure we're ready to convolve
    __syncthreads();

    // perform convolution
    for (k = -half_filter_size; k <= half_filter_size - 1; k++){
        temp_sum += rdata_sh[(threadIdx.x + half_filter_size) - k]*filter_sh[k + half_filter_size];
    }
    // push results to output array
    row_convolved_data[ind] = temp_sum;
}


__global__ void dog_column_convolution(float *data,  float *filter, int rowsize, int colsize, int halfFilt)
{
/*
This function takes input data (row convolved data) and performs a column convolution. The convolution results are
stored in the original input array.

Each row is loaded into shared memory before the convolution is performed. Currently, the maximum size array that can
be convolved by this function is 1024x1024, because each pixel is assigned its own thread.
*/
    int k, halfFiltm1 = halfFilt-1;
    int cid = blockIdx.x;// + halfFilt;
    int j = threadIdx.x;// + halfFilt;
    float tempsum = 0;

    // allocated column of shared memory
    volatile __shared__ float cdata_sh[1075]; //should be changed to rowsize
    __shared__ float filter_sh[12];

    // pad shared mem column
    if (j < (halfFilt)){
        cdata_sh[j] = 0;
        cdata_sh[rowsize + j + halfFilt] = 0;
        //printf("colsize + halfFilt %d", (colsize + halfFilt));
    }
    // load data column into shared mem
    cdata_sh[j + halfFilt] = data[j*colsize + cid];
    // load filter into shared mem
    if (j < (2*halfFilt)) filter_sh[j] = filter[j];

    // make sure we are ready for the convolution
    __syncthreads();

    // perform convolution
    for (k = -halfFilt; k <= halfFiltm1; k++){
        tempsum += cdata_sh[(j + halfFilt) - k]*filter_sh[k + halfFilt];
    }

    // store results in input array
    data[cid + j*colsize] = tempsum;
}

__global__ void weighted_difference_of_gaussian_subtraction(float *uniflargedat, float *uniflargevar, float *unifsmalldat,  float *unifsmallvar,
int colsize, int halfFilt)
{
/*
Perform the necessary divisions and subtractions on previously convolved arrays to achieve the filtering described in
supplemental materials of 10.1038/nmeth.2488.
*/
    int rid = blockIdx.x;
    int cid = threadIdx.x;
    int dloc = rid * colsize + cid;

    uniflargedat[dloc] = (unifsmalldat[dloc]/unifsmallvar[dloc]) - (uniflargedat[dloc]/uniflargevar[dloc]);

}
